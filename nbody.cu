#include "hip/hip_runtime.h"
#include <math.h>
#include <stdio.h>
#include <stdlib.h>
#include "timer.h"

#define SHMOO 1
#define BLOCK_SIZE 256
#define SOFTENING 1e-9f

typedef struct { float x, y, z, vx, vy, vz; } Body;

void randomizeBodies(float *data, int n) {
  for (int i = 0; i < n; i++) {
    data[i] = 2.0f * (rand() / (float)RAND_MAX) - 1.0f;
  }
}

__global__
void bodyForce(Body *p, float dt, int n) {
  int i = blockDim.x * blockIdx.x + threadIdx.x;
  if (i < n) {
    float Fx = 0.0f; float Fy = 0.0f; float Fz = 0.0f;

    for (int j = 0; j < n; j++) {
      float dx = p[j].x - p[i].x;
      float dy = p[j].y - p[i].y;
      float dz = p[j].z - p[i].z;
      float distSqr = dx*dx + dy*dy + dz*dz + SOFTENING;
      float invDist = rsqrtf(distSqr);
      float invDist3 = invDist * invDist * invDist;

      Fx += dx * invDist3; Fy += dy * invDist3; Fz += dz * invDist3;
    }

    p[i].vx += dt*Fx; p[i].vy += dt*Fy; p[i].vz += dt*Fz;
  }
}

int main(const int argc, const char** argv) {

  int nBodies  = 150000;
  int priority = 0;
  if (argc > 1) nBodies  = atoi(argv[1]);
  if (argc > 2) priority = atoi(argv[2]);

  const float dt = 0.01f; // time step
  const int nIters = 20;  // simulation iterations

  int bytes = nBodies*sizeof(Body);
  float *buf = (float*)malloc(bytes);
  Body *p = (Body*)buf;

  randomizeBodies(buf, 6*nBodies); // Init pos / vel data

  float *d_buf;
  hipMalloc(&d_buf, bytes);
  Body *d_p = (Body*)d_buf;

  int nBlocks = (nBodies + BLOCK_SIZE - 1) / BLOCK_SIZE;
  double totalTime = 0.0;

  hipStream_t stream1;
  hipStreamCreateWithPriority(&stream1, hipStreamNonBlocking, priority);
  for (int iter = 1; iter <= nIters; iter++) {
    StartTimer();

    hipMemcpyAsync(d_buf, buf, bytes, hipMemcpyHostToDevice, stream1);
    bodyForce<<<nBlocks, BLOCK_SIZE, 0, stream1>>>(d_p, dt, nBodies); // compute interbody forces
    hipMemcpyAsync(buf, d_buf, bytes, hipMemcpyDeviceToHost, stream1);
    hipDeviceSynchronize();

    for (int i = 0 ; i < nBodies; i++) { // integrate position
      p[i].x += p[i].vx*dt;
      p[i].y += p[i].vy*dt;
      p[i].z += p[i].vz*dt;
    }

    const double tElapsed = GetTimer() / 1000.0;
    if (iter > 1) { // First iter is warm up
      totalTime += tElapsed;
    }
    // printf("Iteration %d: %.3f seconds\n", iter, tElapsed);
  }
  double avgTime = totalTime / (double)(nIters-1);

  // printf("%d, %0.3f\n", nBodies, 1e-9 * nBodies * nBodies / avgTime);
  printf("%d Bodies: average %0.3f Billion Interactions / second\n", nBodies, 1e-9 * nBodies * nBodies / avgTime);
  free(buf);
  hipFree(d_buf);
  hipStreamDestroy(stream1);
}
